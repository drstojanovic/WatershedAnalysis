#include "hip/hip_runtime.h"

//// KERNEL FUNCTIONS IMPLEMENTATION //////////////////////////////////////////////////////////////

#include "Kernels.h"

//// DEPRESSION FILLING ///////////////////////////////////////////////////////////////////////////

__global__ void DepressionFillingKernel(
	float* wDEM,
	float* zDEM,
	int width,
	int height,
	float epsilon,
	bool* pFinished)
{
	int row = blockIdx.y * O_BLOCK_WIDTH + threadIdx.y + 1;
	int col = blockIdx.x * O_BLOCK_WIDTH + threadIdx.x + 1;

	int row_i = row - 1;
	int col_i = col - 1;

	int row_s = threadIdx.y + 1;
	int col_s = threadIdx.x + 1;

	__shared__ float ds_wDEM[I_BLOCK_WIDTH][I_BLOCK_WIDTH];

	int center;

	float zDEMCenter;
	float wDEMCenter;
	float wDEMNeighbor;


	if (row_i < height && col_i < width)
	{
		ds_wDEM[threadIdx.y][threadIdx.x] = wDEM[row_i * width + col_i];
	}

	__syncthreads();

	if (row < height - 1 && col < width - 1 && threadIdx.y < O_BLOCK_WIDTH && threadIdx.x < O_BLOCK_WIDTH)
	{
		center = row * width + col;

		zDEMCenter = zDEM[center];
		wDEMCenter = ds_wDEM[row_s][col_s];

		if (wDEMCenter > zDEMCenter)
		{
			for (int k = row_s - 1; k <= row_s + 1; ++k)
			{
				for (int l = col_s - 1; l <= col_s + 1; ++l)
				{
					if (k != row_s || l != col_s)
					{
						wDEMNeighbor = ds_wDEM[k][l] + epsilon;

						if (zDEMCenter >= wDEMNeighbor)
						{
							wDEMCenter = zDEMCenter;
							ds_wDEM[row_s][col_s] = zDEMCenter;
							*pFinished = false;
						}
						else
						{
							if (wDEMCenter > wDEMNeighbor)
							{
								wDEMCenter = wDEMNeighbor;
								ds_wDEM[row_s][col_s] = wDEMNeighbor;
								*pFinished = false;
							}
						}
					}
				}
			}

			wDEM[center] = wDEMCenter;
		}
	}
}

void DepressionFillingKernel_Wrapper(dim3 dimGrid, dim3 dimBlock,
	float* wDEM,
	float* zDEM,
	int width,
	int height,
	float epsilon,
	bool* pFinished)
{
	DepressionFillingKernel<<<dimGrid, dimBlock>>>
		(wDEM, zDEM, width, height, epsilon, pFinished);
}

//// FLOW-TRANSFER-MATRIX FLOW ACCUMULATION ///////////////////////////////////////////////////////

__global__ void FlowAccumulationKernelM(
	float* flowOld,
	float* flowNew,
	float* flowAcc,
	float* flowFracs,
	unsigned char* flowRevDirs,
	int width,
	int height,
	bool* pFinished)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float newFlow;
	unsigned char flowRevDir;

	int center;
	int stridedCenter;

	int subCenterWidth;
	int addCenterWidth;

	if (row < height && col < width)
	{
		center = row * width + col;
		stridedCenter = row * width * 8 + col;

		subCenterWidth = center - width;
		addCenterWidth = center + width;

		flowRevDir = flowRevDirs[center];
		newFlow = 0.0f;

		if (flowRevDir & 1)   newFlow += flowFracs[stridedCenter] * flowOld[subCenterWidth];
		stridedCenter += width;

		if (flowRevDir & 2)   newFlow += flowFracs[stridedCenter] * flowOld[subCenterWidth + 1];
		stridedCenter += width;

		if (flowRevDir & 4)   newFlow += flowFracs[stridedCenter] * flowOld[center + 1];
		stridedCenter += width;

		if (flowRevDir & 8)   newFlow += flowFracs[stridedCenter] * flowOld[addCenterWidth + 1];
		stridedCenter += width;

		if (flowRevDir & 16)  newFlow += flowFracs[stridedCenter] * flowOld[addCenterWidth];
		stridedCenter += width;

		if (flowRevDir & 32)  newFlow += flowFracs[stridedCenter] * flowOld[addCenterWidth - 1];
		stridedCenter += width;

		if (flowRevDir & 64)  newFlow += flowFracs[stridedCenter] * flowOld[center - 1];
		stridedCenter += width;

		if (flowRevDir & 128) newFlow += flowFracs[stridedCenter] * flowOld[subCenterWidth - 1];


		flowAcc[center] += newFlow;
		flowNew[center] = newFlow;

		if (newFlow > 0.0f)
		{
			*pFinished = false;
		}
	}
}

void FlowAccumulationKernelM_Wrapper(dim3 dimGrid, dim3 dimBlock,
	float* flowOld,
	float* flowNew,
	float* flowAcc,
	float* flowFracs,
	unsigned char* flowRevDirs,
	int width,
	int height,
	bool* pFinished)
{
	FlowAccumulationKernelM<<<dimGrid, dimBlock>>>
		(flowOld, flowNew, flowAcc, flowFracs, flowRevDirs, width, height, pFinished);
}

//// GRAPH-BASED FLOW ACCUMULATION ////////////////////////////////////////////////////////////////

__global__ void FlowAccumulationKernelG(
	float* flowAcc,
	float* flowFracs,
	unsigned char* flowDirs,
	unsigned char* flowRevDirs,
	int* inDegreeMatrix,
	int width,
	int height,
	bool* pFinished)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int center;
	int stridedCenter;

	int subCenterWidth;
	int addCenterWidth;

	float newFlow;
	unsigned char dir;


	if (row < height && col < width)
	{
		center = row * width + col;

		if (inDegreeMatrix[center] == 0)
		{
			inDegreeMatrix[center] = -1;
			*pFinished = false;

			subCenterWidth = center - width;
			addCenterWidth = center + width;

			stridedCenter = row * width * 8 + col;
			newFlow = 0.0f;

			dir = flowRevDirs[center];

			if (dir & 1)   newFlow += flowFracs[stridedCenter] * flowAcc[subCenterWidth];
			stridedCenter += width;

			if (dir & 2)   newFlow += flowFracs[stridedCenter] * flowAcc[subCenterWidth + 1];
			stridedCenter += width;

			if (dir & 4)   newFlow += flowFracs[stridedCenter] * flowAcc[center + 1];
			stridedCenter += width;

			if (dir & 8)   newFlow += flowFracs[stridedCenter] * flowAcc[addCenterWidth + 1];
			stridedCenter += width;

			if (dir & 16)  newFlow += flowFracs[stridedCenter] * flowAcc[addCenterWidth];
			stridedCenter += width;

			if (dir & 32)  newFlow += flowFracs[stridedCenter] * flowAcc[addCenterWidth - 1];
			stridedCenter += width;

			if (dir & 64)  newFlow += flowFracs[stridedCenter] * flowAcc[center - 1];
			stridedCenter += width;

			if (dir & 128) newFlow += flowFracs[stridedCenter] * flowAcc[subCenterWidth - 1];

			flowAcc[center] += newFlow;

			dir = flowDirs[center];

			if (dir & 1)   atomicSub(&inDegreeMatrix[subCenterWidth],     1);
			if (dir & 2)   atomicSub(&inDegreeMatrix[subCenterWidth + 1], 1);
			if (dir & 4)   atomicSub(&inDegreeMatrix[center + 1],         1);
			if (dir & 8)   atomicSub(&inDegreeMatrix[addCenterWidth + 1], 1);
			if (dir & 16)  atomicSub(&inDegreeMatrix[addCenterWidth],     1);
			if (dir & 32)  atomicSub(&inDegreeMatrix[addCenterWidth - 1], 1);
			if (dir & 64)  atomicSub(&inDegreeMatrix[center - 1],         1);
			if (dir & 128) atomicSub(&inDegreeMatrix[subCenterWidth - 1], 1);
		}
	}
}

void FlowAccumulationKernelG_Wrapper(dim3 dimGrid, dim3 dimBlock,
	float* flowAcc,
	float* flowFracs,
	unsigned char* flowDirs,
	unsigned char* flowRevDirs,
	int* inDegreeMatrix,
	int width,
	int height,
	bool* pFinished)
{

	FlowAccumulationKernelG<<<dimGrid, dimBlock>>>
		(flowAcc, flowFracs, flowDirs, flowRevDirs, inDegreeMatrix, width, height, pFinished);

	
}

///////////////////////////////////////////////////////////////////////////////////////////////////



